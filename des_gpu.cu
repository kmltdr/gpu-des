#include "des_gpu.cuh"
#include "des_kernels.cuh"
#include "des_cpu.h"
#include <cstdint>

#include <hip/hip_runtime_api.h>

uint64_t des_gpu_crack(uint64_t message, uint64_t cipher, uint64_t start)
{
    bool h_done, *d_done;
    checkCudaErrors(hipMalloc((void**)&d_done,sizeof(bool)));
    checkCudaErrors(hipMemset(d_done, 0, sizeof(bool)));
    h_done = false;
    uint64_t h_key = 0, *d_key;
    checkCudaErrors(hipMalloc((void**)&d_key,sizeof(uint64_t)));

    const size_t num_of_blocks = 1024;
    const size_t block_size = 512;
    //while(!h_done)
    //{
        des_gpu_crack_kernel<<<num_of_blocks, block_size>>>(message, cipher,
                start,1000000000, d_done, d_key);
        checkCudaErrors(hipMemcpy(&h_done, d_done, sizeof(bool), hipMemcpyDeviceToHost));
    //    start += num_of_blocks * block_size;
    //}
    if(!h_done) return 0;
    checkCudaErrors(hipMemcpy(&h_key, d_key, sizeof(h_key), hipMemcpyDeviceToHost));
	return des_cpu::rev_permute_add_parity(h_key);
}
